#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_matmul.h"

static void HandleError(hipError_t err, const char* file, int line) {
  // CUDA error handeling from the "CUDA by example" book
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void matrixMultiply_32hd_Kernel(double* v_A, double* v_B,
                                           double* v_C, int A_row, int A_col,
                                           int B_col, int stride,
                                           int num_eqations) {
  extern __shared__ double sums[];

  int thread_idx = threadIdx.x;
  int equ_idx = blockIdx.x;

  Eigen::Map<Eigen::MatrixXd> d_A(v_A + equ_idx * A_row * A_col, A_row, A_col);
  Eigen::Map<Eigen::VectorXd> d_B(v_B + equ_idx * A_col * B_col, A_col, B_col);
  Eigen::Map<Eigen::MatrixXd> d_C(v_C + equ_idx * A_row * B_col, A_row, B_col);

  for (int k = 0; k < B_col; k++) {
    for (int j = 0; j < A_col; j++) {
      for (int i = 0; i < stride; i++) {
        int row = i * 32 + thread_idx;
        int col = j;

        if (row < A_row && col < A_col) {
          if (col == 0) {
            sums[row] = 0.0;
          }

          if (row < A_row) {
            sums[row] += d_A(row, col) * d_B(col, k);
          }

          if (col == A_col - 1) {
            d_C(row, k) = sums[row];
          }
        }
      }
    }
  }
}

void matrixMultiply_32thd(std::vector<Eigen::MatrixXd>& v_A,
                          std::vector<Eigen::MatrixXd>& v_B,
                          std::vector<Eigen::MatrixXd>& v_C,
                          int num_equations) {
  int M = v_A[0].rows();
  int N = v_A[0].cols();
  int K = v_B[0].cols();

  size_t size_vA = num_equations * M * N * sizeof(double);
  size_t size_vB = num_equations * N * K * sizeof(double);
  size_t size_vC = num_equations * M * K * sizeof(double);

  double* d_vA;
  double* d_vB;
  double* d_vC;

  // Allocate device memory
  HANDLE_ERROR(hipMalloc((void**)&d_vA, size_vA));
  HANDLE_ERROR(hipMalloc((void**)&d_vB, size_vB));
  HANDLE_ERROR(hipMalloc((void**)&d_vC, size_vC));

  // Copy data to device
  for (int i = 0; i < num_equations; i++) {
    HANDLE_ERROR(hipMemcpy(d_vA + i * M * N, v_A[i].data(),
                            M * N * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_vB + i * N * K, v_B[i].data(),
                            N * K * sizeof(double), hipMemcpyHostToDevice));
  }

  // Define block and grid sizes
  int threadsPerBlock = 32;
  int numBlocks = num_equations;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  int stride = (M + threadsPerBlock - 1) / threadsPerBlock;
  // Launch kernel
  matrixMultiply_32hd_Kernel<<<numBlocks, threadsPerBlock,
                               2048 * sizeof(double)>>>(
      d_vA, d_vB, d_vC, M, N, K, stride, num_equations);
  hipDeviceSynchronize();

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for matrixMultiply_32hd_Kernel: " << milliseconds
            << " ms\n";

  // Copy result back to host
  for (int i = 0; i < num_equations; i++) {
    HANDLE_ERROR(hipMemcpy(v_C[i].data(), d_vC + i * M * K,
                            M * K * sizeof(double), hipMemcpyDeviceToHost));
  }

  // Free device memory
  HANDLE_ERROR(hipFree(d_vA));
  HANDLE_ERROR(hipFree(d_vB));
  HANDLE_ERROR(hipFree(d_vC));
}