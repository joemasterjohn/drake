#include "hip/hip_runtime.h"
#include <stdio.h>

#include <iostream>

#include "cuda_gauss_jordan.h"

static void HandleError(hipError_t err, const char* file, int line) {
  // CUDA error handeling from the "CUDA by example" book
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void gauss_jordan_inverse(double* M, double* inv,
                                     size_t num_equations, size_t n,
                                     size_t offset) {
  int equ_idx = blockIdx.x;
  int thread_idx = threadIdx.x;

  if (equ_idx >= num_equations) {
    return;
  }

  Eigen::Map<Eigen::MatrixXd> d_M(M + equ_idx * n * n, n, n);
  Eigen::Map<Eigen::MatrixXd> d_inv(inv + equ_idx * n * n, n, n);

  size_t row = thread_idx + offset;

  if (row >= n || row == 0) {
    return;
  }

  for (size_t i = 0; i < 32; i++) {
    size_t pivot = i + offset;
    double mult = d_M(pivot, pivot) / d_M(row, pivot);
    for (size_t j = pivot; j < n; j++) {
      if (row > pivot) {
        d_M(row, j) = d_M(pivot, pivot) - mult * d_M(row, j);
        d_inv(row, j) = d_inv(pivot, pivot) - mult * d_inv(row, j);
      }
      __syncwarp();
    }
    __syncwarp();
  }

  // for (i = matrix_size − 1; i > 0; i−−) {
  //   for (y = 0; y < i; y++) {
  //     mult = matrix[i][i] / matrix[y][i];
  //     for (x = 0; x < i + 1; x++)
  //       13 matrix[y][x] = matrix[i][i] − mult ∗ matrix[y][x];
  //   }
}

double gauss_jordan_solve(std::vector<Eigen::MatrixXd>& M,
                          std::vector<Eigen::MatrixXd>& I) {
  const int num_equations = M.size();
  const int n = M[0].rows();

  // Allocate device arrays
  double *d_M, *d_inv;
  HANDLE_ERROR(
      hipMalloc((void**)&d_M, sizeof(double) * num_equations * n * n));
  HANDLE_ERROR(
      hipMalloc((void**)&d_inv, sizeof(double) * num_equations * n * n));

  // Copy to device
  for (int i = 0; i < num_equations; ++i) {
    HANDLE_ERROR(hipMemcpy(d_M + i * n * n, M[i].data(),
                            sizeof(double) * n * n, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_inv + i * n * n, I[i].data(),
                            sizeof(double) * n * n, hipMemcpyHostToDevice));
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  int offset = 0;
  while (offset < n) {
    gauss_jordan_inverse<<<num_equations, 32>>>(d_M, d_inv, num_equations, n,
                                                offset);
    offset += 32;

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
  }

  hipEventRecord(stop);

  hipEventSynchronize(stop);
  float milliseconds = 0.0;
  hipEventElapsedTime(&milliseconds, start, stop);

  std::cout << "Elapsed time for Gauss Jordan Inverse: " << milliseconds
            << " ms\n";

  // Copy to host
  // HANDLE_ERROR(hipMemcpy(M.data, d_M, sizeof(double) * num_equations * n *
  // n,
  //                         hipMemcpyDeviceToHost));  // just for testing
  for (int i = 0; i < num_equations; i++) {
    HANDLE_ERROR(hipMemcpy(I[i].data(), d_inv + i * n * n,
                            sizeof(double) * num_equations * n * n,
                            hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(M[i].data(), d_M + i * n * n,
                            sizeof(double) * num_equations * n * n,
                            hipMemcpyDeviceToHost));
  }

  for (int i = 0; i < num_equations; ++i) {
    std::cout << M[i] << std::endl;
    // std::cout << "inverse err: " << (M[i].inverse() - x_result_i).norm()
    //           << std::endl;
  }

  return 0;
}
